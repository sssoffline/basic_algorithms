
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>

__global__ void staticReverse(int *d, int n)
{
  __shared__ int s[64];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

__global__ void dynamicReverse(int *d, int n)
{
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main(void)
{
  const int n = 64;
  int a[n], r[n], d[n];

  for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n-i-1;
    d[i] = 0;
  }

  int *d_d;
  hipMalloc(&d_d, n * sizeof(int));

  // run version with static shared memory
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  printf("running static reverse ....\n");
  staticReverse<<<1,n>>>(d_d, n);
  hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);

  // run dynamic shared memory version
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  printf("running dynamic reverse ....\n");
  dynamicReverse<<<1,n,n*sizeof(int)>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++)
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);

  printf("complete execution.\n");
}
